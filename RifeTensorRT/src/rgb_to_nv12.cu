#include "hip/hip_runtime.h"
// color_conversion.cu

#include <hip/hip_runtime.h>
#include <stdint.h>
#include <stdio.h>
#include <hip/hip_fp16.h> // Include for half precision

// Existing CUDA_CHECK macro
#define CUDA_CHECK(err) \
    if (err != hipSuccess) { \
        fprintf(stderr, "CUDA Error: %s (err_num=%d)\n", hipGetErrorString(err), err); \
        exit(err); \
    }

extern "C" {

    // FP32 Kernel for RGB to NV12 conversion
    __global__ void rgb_to_nv12_fp32(const float* __restrict__ rgb, uint8_t* y, uint8_t* uv,
        int width, int height, int y_linesize, int uv_linesize) {
        int x = blockIdx.x * blockDim.x + threadIdx.x;
        int y_pos = blockIdx.y * blockDim.y + threadIdx.y;

        // Ensure we don't go out of bounds.
        if (x >= width || y_pos >= height) return;

        // Calculate the index for RGB data in channels-first format (C, H, W)
        int idx = y_pos * width + x;

        // Load RGB values directly from the float data.
        float r_f = rgb[idx]; // Red
        float g_f = rgb[width * height + idx]; // Green
        float b_f = rgb[2 * width * height + idx]; // Blue

        // Normalize by multiplying by 255.
        r_f *= 255.0f;
        g_f *= 255.0f;
        b_f *= 255.0f;

        // Clamp values to [0, 255].
        r_f = fminf(fmaxf(r_f, 0.0f), 255.0f);
        g_f = fminf(fmaxf(g_f, 0.0f), 255.0f);
        b_f = fminf(fmaxf(b_f, 0.0f), 255.0f);

        // Convert to uint8 for output.
        uint8_t r = static_cast<uint8_t>(r_f);
        uint8_t g = static_cast<uint8_t>(g_f);
        uint8_t b = static_cast<uint8_t>(b_f);

        // Convert RGB to Y (luminance).
        y[y_pos * y_linesize + x] = static_cast<uint8_t>(0.299f * r + 0.587f * g + 0.114f * b);

        // For U and V, subsample and interleave them into NV12 format.
        if (y_pos % 2 == 0 && x % 2 == 0) {
            int sub_idx = (y_pos / 2) * (width / 2) + (x / 2);

            // Calculate U and V (chrominance).
            float u_f = (-0.14713f * r - 0.28886f * g + 0.436f * b + 128.0f);
            float v_f = (0.615f * r - 0.51499f * g - 0.10001f * b + 128.0f);

            // Clamp U and V to [0, 255].
            u_f = fminf(fmaxf(u_f, 0.0f), 255.0f);
            v_f = fminf(fmaxf(v_f, 0.0f), 255.0f);

            uint8_t u = static_cast<uint8_t>(u_f);
            uint8_t v = static_cast<uint8_t>(v_f);

            // Interleave U and V in NV12 format.
            uv[(y_pos / 2) * uv_linesize + x] = u;
            uv[(y_pos / 2) * uv_linesize + x + 1] = v;
        }
    }

    // FP16 Kernel for RGB to NV12 conversion
    __global__ void rgb_to_nv12_fp16(const __half* __restrict__ rgb, uint8_t* y, uint8_t* uv,
        int width, int height, int y_linesize, int uv_linesize) {
        int x = blockIdx.x * blockDim.x + threadIdx.x;
        int y_pos = blockIdx.y * blockDim.y + threadIdx.y;

        // Ensure we don't go out of bounds.
        if (x >= width || y_pos >= height) return;

        // Calculate the index for RGB data in channels-first format (C, H, W)
        int idx = y_pos * width + x;

        // Load RGB values directly from the half data.
        __half r_h = rgb[idx]; // Red
        __half g_h = rgb[width * height + idx]; // Green
        __half b_h = rgb[2 * width * height + idx]; // Blue

        // Convert half to float for processing.
        float r_f = __half2float(r_h) * 255.0f;
        float g_f = __half2float(g_h) * 255.0f;
        float b_f = __half2float(b_h) * 255.0f;

        // Clamp values to [0, 255].
        r_f = fminf(fmaxf(r_f, 0.0f), 255.0f);
        g_f = fminf(fmaxf(g_f, 0.0f), 255.0f);
        b_f = fminf(fmaxf(b_f, 0.0f), 255.0f);

        // Convert to uint8 for output.
        uint8_t r = static_cast<uint8_t>(r_f);
        uint8_t g = static_cast<uint8_t>(g_f);
        uint8_t b = static_cast<uint8_t>(b_f);

        // Convert RGB to Y (luminance).
        y[y_pos * y_linesize + x] = static_cast<uint8_t>(0.299f * r + 0.587f * g + 0.114f * b);

        // For U and V, subsample and interleave them into NV12 format.
        if (y_pos % 2 == 0 && x % 2 == 0) {
            int sub_idx = (y_pos / 2) * (width / 2) + (x / 2);

            // Calculate U and V (chrominance).
            float u_f = (-0.14713f * r - 0.28886f * g + 0.436f * b + 128.0f);
            float v_f = (0.615f * r - 0.51499f * g - 0.10001f * b + 128.0f);

            // Clamp U and V to [0, 255].
            u_f = fminf(fmaxf(u_f, 0.0f), 255.0f);
            v_f = fminf(fmaxf(v_f, 0.0f), 255.0f);

            uint8_t u = static_cast<uint8_t>(u_f);
            uint8_t v = static_cast<uint8_t>(v_f);

            // Interleave U and V in NV12 format.
            uv[(y_pos / 2) * uv_linesize + x] = u;
            uv[(y_pos / 2) * uv_linesize + x + 1] = v;
        }
    }

    // FP32 Kernel Launcher
    void launch_rgb_to_nv12_fp32(const float* rgb, uint8_t* y, uint8_t* uv,
        int width, int height, int y_linesize, int uv_linesize,
        hipStream_t stream) {
        dim3 threads(16, 16);  // Threads per block
        dim3 blocks((width + threads.x - 1) / threads.x, (height + threads.y - 1) / threads.y);  // Number of blocks

        // Launch the FP32 kernel
        rgb_to_nv12_fp32 << <blocks, threads, 0, stream >> > (rgb, y, uv, width, height, y_linesize, uv_linesize);

        // Check for kernel launch errors.
        hipError_t err = hipGetLastError();
        CUDA_CHECK(err);
    }

    // FP16 Kernel Launcher
    void launch_rgb_to_nv12_fp16(const __half* rgb, uint8_t* y, uint8_t* uv,
        int width, int height, int y_linesize, int uv_linesize,
        hipStream_t stream) {
        dim3 threads(16, 16);  // Threads per block
        dim3 blocks((width + threads.x - 1) / threads.x, (height + threads.y - 1) / threads.y);  // Number of blocks

        // Launch the FP16 kernel
        rgb_to_nv12_fp16 << <blocks, threads, 0, stream >> > (rgb, y, uv, width, height, y_linesize, uv_linesize);

        // Check for kernel launch errors.
        hipError_t err = hipGetLastError();
        CUDA_CHECK(err);
    }

    // Similarly, implement nv12_to_rgb_fp32 and nv12_to_rgb_fp16 if needed.

}  // extern "C"
